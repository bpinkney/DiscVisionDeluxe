#include "hip/hip_runtime.h"
#include<iostream>
#include<limits>
#include <chrono>
#include <unordered_map>
#include <string>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// GPU version
//#include "nvAprilTags.h"
//#include "apriltag_gpu.h"

// CPU version
#include "apriltag.h"
#include "tag36h11.h"

// opencv stuff
#include "opencv2/core.hpp"
#include <opencv2/core/utility.hpp>
#include "opencv2/imgproc.hpp"
#include "opencv2/calib3d.hpp"
#include "opencv2/imgcodecs.hpp"
#include "opencv2/videoio.hpp"
#include "opencv2/highgui.hpp"

using namespace std;
using namespace cv;

static nvAprilTagsHandle hApriltags = nullptr;

// camera matrix is 3x3:
//  fx   0    cx
//  0    fy   cy
//  0    0    1 

// [ 1.5283289799775675e+03, 0., 3.5971937738154656e+02, 
//   0., 1.4032086558666635e+03, 2.6986410577371180e+02, 
//   0., 0., 1. ]

static float sFx = 1.5283289799775675e+03;
static float sFy = 1.4032086558666635e+03;
static float sCx = 3.5971937738154656e+02;
static float sCy = 2.6986410577371180e+02;
static float sTagDimension = 0.20;
static uint8_t* buffer_cpu = nullptr;
static uint8_t* buffer_gpu = nullptr;

nvAprilTagsImageInput_st sNvData;


void allocate_uint8_buffer(uint8_t *tempPoint, const int width, const int height, const int color_comp_per_pixel)
{
  //uint8_t *tempPoint;// = nullptr;
  const uint32_t gps_malloc_size = width*height*color_comp_per_pixel*sizeof(uint8_t);

  // Why does this keep segfaulting?
  checkCudaErrors(hipMalloc((void **)&tempPoint, gps_malloc_size));
  //return tempPoint;
}

int release_gpu_buffer (void * buffer)
{
  checkCudaErrors(hipFree(buffer));
  return 0;
}


int upload_data_to_GPU (void * dst, void * src, int width, int height, int color_comp_per_pixel)
{
  int size = width * height * color_comp_per_pixel;    
  checkCudaErrors(hipMemcpy(dst,src,size,hipMemcpyHostToDevice));
  return 0;
}

int download_data_from_GPU (void * dst, void * src, int width, int height, int color_comp_per_pixel)
{
  int size = width * height * color_comp_per_pixel;        
  checkCudaErrors(hipMemcpy(dst,src,size,hipMemcpyDeviceToHost));
  return 0;
}

void upload_data_to_uchar4_gpu(uint8_t *data, int pitch, int width, int height) 
{

  uchar4* buf = (uchar4 *)buffer_cpu;
  int i = 0, j = 0;
  if(IMAGE_C == 4) {
    while(i < pitch * height * IMAGE_C) {
      buf[j].x = data[i++];
      buf[j].y = data[i++];
      buf[j].z = data[i++];
      buf[j].w = data[i++];
      j++;
    }
  } else {
    while(i < pitch * height * IMAGE_C) {
      buf[j].x = data[i];
      buf[j].y = data[i];
      buf[j].z = data[i];
      buf[j].w = 0xFF; //data[i];
      j++; i++;
    }
  }
  upload_data_to_GPU(buffer_gpu, buffer_cpu, pitch, height, sizeof(uchar4));
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
int nv_init_apriltag_detector() 
{
  if(hApriltags) return -1;
  buffer_cpu = (uint8_t*)malloc(sizeof(uchar4) * IMAGE_W * IMAGE_H);
  //buffer_gpu = (uint8_t*)malloc(sizeof(uchar4) * IMAGE_W * IMAGE_H);
  allocate_uint8_buffer(buffer_gpu, IMAGE_W, IMAGE_H, sizeof(uchar4));
  return 0;
}

int nv_create_apriltag_detector(int width, int height) 
{
  if(hApriltags) nv_destroy_apriltag_detector();

  nvAprilTagsCameraIntrinsics_t cam = {sFx, sFy, sCx, sCy };
  sNvData.dev_ptr = (uchar4*)buffer_gpu;
  sNvData.pitch   = width * sizeof(uchar4);
  sNvData.width   = width;
  sNvData.height  = height;
  return nvCreateAprilTagsDetector(&hApriltags, width, height, NVAT_TAG36H11, &cam, sTagDimension);
}

static unordered_map<int, int> m;

int nv_apriltag_detector_detect (uint8_t *data, int pitch, int width, int height) 
{
  nv_create_apriltag_detector(width, height);

  if(!hApriltags) return -1;
  
  upload_data_to_uchar4_gpu(data, pitch, width, height);

  uint32_t num_tags = 0;
  std::vector<nvAprilTagsID_t> tags;

  auto prev = std::chrono::high_resolution_clock::now();

  sNvData.pitch   = pitch * sizeof(uchar4);
  sNvData.width   = width;
  sNvData.height  = height;

  // for(int i = 0; i < 1000; i++) 
  {
    tags.clear();
    tags.resize(20);
    nvAprilTagsDetect(hApriltags, &(sNvData), tags.data(), &num_tags, tags.size(), nullptr);
  }
  auto now = std::chrono::high_resolution_clock::now();

  std::chrono::duration<double, std::milli> elapse_ms = now - prev;

  // std::cout << "delay = " << elapse_ms.count() << " ms\n";

  for(int i = 0 ; i < num_tags; i++) {
    cout << "tag id " << tags[i].id << "\n";
    m[tags[i].id]++;
  }
  return num_tags;
}


void nv_destroy_apriltag_detector() 
{
  if(!hApriltags) return;
  nvAprilTagsDestroy(hApriltags);
}



void nv_print_stats() 
{
  vector<int> ids = {173, 319, 119, 219, 142};
  for(auto id : ids) 
  {
    std::cout << "id = " << id << " count " << m[id] << "\n";
  }
}

#define IMAGE_C (1.0)
int main( int argc, char** argv )
{
  std::cout << "TEST" << endl;

  // Load image
  std::string folderpath = "undistorted_imgs/*.jpg";
  // get list of images
  vector<std::string> filenames;
  glob(folderpath, filenames);

  Mat view, viewGray = imread(filenames[0], 1);
  Size imageSize = view.size();

  // aprilTag Stuff
  const int total_size = IMAGE_W * IMAGE_H * IMAGE_C;
  uint8_t *image_buffer = new uint8_t[total_size];

  //nv_init_apriltag_detector();

  int total_detections = 0;

  int i;
  for( i = 0; i < (int)filenames.size(); i++ )
  {
      view = imread(filenames[i], 1);
      if(view.empty())
          continue;

      int im_height = view.rows;
      int im_width  = view.cols;

      // convert to greyscale
      cvtColor(view, viewGray, cv::COLOR_RGB2GRAY);

      std::memcpy(image_buffer, viewGray.data, viewGray.total() * sizeof(uint8_t));
      
      int det1 = nv_apriltag_detector_detect(image_buffer, im_width, im_width, im_height);
      total_detections += det1;

      imshow("Image View", viewGray);

      char c = (char)waitKey();
      if( c == 27 || c == 'q' || c == 'Q' )
          break;
  }

  nv_destroy_apriltag_detector();  
}
