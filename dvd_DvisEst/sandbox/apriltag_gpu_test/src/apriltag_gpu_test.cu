#include "hip/hip_runtime.h"
#include<iostream>
#include<limits>
#include <chrono>
#include <unordered_map>
#include "nvAprilTags.h"

//#include "apriltag_gpu.h"

using namespace std;

static nvAprilTagsHandle hApriltags = nullptr;

static float sFx = 205;
static float sFy = 205;
static float sCx = 320;
static float sCy = 240;
static float sTagDimension = 0.224;
static uint8_t* buffer_cpu = nullptr;
static uint8_t* buffer_gpu = nullptr;

nvAprilTagsImageInput_st sNvData;


uint8_t *allocate_uint8_buffer(int width, int height, int color_comp_per_pixel)
{
    uint8_t *tempPoint = nullptr;
    checkCudaErrors(hipMalloc((void **)&tempPoint, width*height*color_comp_per_pixel*sizeof(uint8_t) ));
    return tempPoint;
}

int release_gpu_buffer (void * buffer)
{
    checkCudaErrors(hipFree(buffer));
    return 0;
}


int upload_data_to_GPU (void * dst, void * src, int width, int height, int color_comp_per_pixel)
{
    int size = width * height * color_comp_per_pixel;    
    checkCudaErrors(hipMemcpy(dst,src,size,hipMemcpyHostToDevice));
    return 0;
}

int download_data_from_GPU (void * dst, void * src, int width, int height, int color_comp_per_pixel)
{
    int size = width * height * color_comp_per_pixel;        
    checkCudaErrors(hipMemcpy(dst,src,size,hipMemcpyDeviceToHost));
    return 0;
}

void upload_data_to_uchar4_gpu(uint8_t *data, int pitch, int width, int height) {

    uchar4* buf = (uchar4 *)buffer_cpu;
    int i = 0, j = 0;
    if(IMAGE_C == 4) {
        while(i < pitch * height * IMAGE_C) {
            buf[j].x = data[i++];
            buf[j].y = data[i++];
            buf[j].z = data[i++];
            buf[j].w = data[i++];
            j++;
        }
    } else {
        while(i < pitch * height * IMAGE_C) {
            buf[j].x = data[i];
            buf[j].y = data[i];
            buf[j].z = data[i];
            buf[j].w = 0xFF; //data[i];
            j++; i++;
        }
    }
    upload_data_to_GPU(buffer_gpu, buffer_cpu, pitch, height, sizeof(uchar4));
}

///////////////////////////////////////////////////////////////////////////////////////////////////////////
int nv_init_apriltag_detector() {
    if(hApriltags) return -1;
    buffer_cpu = (uint8_t*)malloc(sizeof(uchar4) * IMAGE_W * IMAGE_H);
    buffer_gpu = allocate_uint8_buffer(IMAGE_W, IMAGE_H, sizeof(uchar4));
    return 0;
}

int nv_create_apriltag_detector(int width, int height) {
    if(hApriltags) nv_destroy_apriltag_detector();

    nvAprilTagsCameraIntrinsics_t cam = {sFx, sFy, sCx, sCy };
    sNvData.dev_ptr = (uchar4*)buffer_gpu;
    sNvData.pitch   = width * sizeof(uchar4);
    sNvData.width   = width;
    sNvData.height  = height;
    return nvCreateAprilTagsDetector(&hApriltags, width, height, NVAT_TAG36H11, &cam, sTagDimension);
}

static unordered_map<int, int> m;

int nv_apriltag_detector_detect (uint8_t *data, int pitch, int width, int height) {
    nv_create_apriltag_detector(width, height);

    if(!hApriltags) return -1;
    
    upload_data_to_uchar4_gpu(data, pitch, width, height);

    uint32_t num_tags = 0;
    std::vector<nvAprilTagsID_t> tags;

    auto prev = std::chrono::high_resolution_clock::now();

    sNvData.pitch   = pitch * sizeof(uchar4);
    sNvData.width   = width;
    sNvData.height  = height;

    // for(int i = 0; i < 1000; i++) 
    {
        tags.clear();
        tags.resize(20);
        nvAprilTagsDetect(hApriltags, &(sNvData), tags.data(), &num_tags, tags.size(), nullptr);
    }
    auto now = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double, std::milli> elapse_ms = now - prev;

    // std::cout << "delay = " << elapse_ms.count() << " ms\n";

    for(int i = 0 ; i < num_tags; i++) {
        cout << "tag id " << tags[i].id << "\n";
        m[tags[i].id]++;
    }
    return num_tags;
}


void nv_destroy_apriltag_detector() {
    if(!hApriltags) return;
    nvAprilTagsDestroy(hApriltags);
}



void nv_print_stats() {
    vector<int> ids = {173, 319, 119, 219, 142};
    for(auto id : ids) {
        std::cout << "id = " << id << " count " << m[id] << "\n";
    }
}
